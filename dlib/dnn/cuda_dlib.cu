#include "hip/hip_runtime.h"
// Copyright (C) 2015  Davis E. King (davis@dlib.net)
// License: Boost Software License   See LICENSE.txt for the full license.

#include "cuda_utils.h"
#include "cuda_dlib.h"


namespace dlib 
{ 
    namespace cuda 
    {

    // -----------------------------------------------------------------------------------

        void set_device (
            int dev
        )
        {
            CHECK_CUDA(hipSetDevice(dev));
        }

        int get_device (
        )
        {
            int dev = 0;
            CHECK_CUDA(hipGetDevice(&dev));
            return dev;
        }

        int get_num_devices (
        )
        {
            int num_devices;
            CHECK_CUDA(hipGetDeviceCount(&num_devices));
            return num_devices;
        }

        bool can_access_peer (int device_id, int peer_device_id)
        {
            int can_access;
            CHECK_CUDA(hipDeviceCanAccessPeer(&can_access, device_id, peer_device_id));
            return can_access;
        }
        bool can_access_peer (const tensor& device, const tensor& peer_device)
        {
            return can_access_peer(device.device_id(), peer_device.device_id());
        }

        void device_synchronize (int dev) 
        { 
            raii_set_device set_dev(dev);
            CHECK_CUDA(hipDeviceSynchronize());
        }
        void device_synchronize (const tensor& dev) { device_synchronize(dev.device_id()); }

        enable_peer_access::
        enable_peer_access(
            int device_id,
            int peer_device_id
        ) : call_disable(false), device_id(device_id), peer_device_id(peer_device_id)
        {
            raii_set_device set_dev(device_id);

            auto err = hipDeviceEnablePeerAccess(peer_device_id, 0);
            if (err == hipSuccess)
            {
                call_disable = true;
            }
            else if (err == hipErrorPeerAccessAlreadyEnabled)
            {
                // call hipGetLastError() to dispose of this error since we don't
                // care.
                auto err2 = hipGetLastError();
                if (err2 != hipErrorPeerAccessAlreadyEnabled)
                    CHECK_CUDA(err2);
            }
            else
            {
                CHECK_CUDA(err);
            }
        }


        enable_peer_access::
        ~enable_peer_access() noexcept(false)
        {
            if (call_disable)
            {
                raii_set_device set_dev(device_id);
                CHECK_CUDA(hipDeviceDisablePeerAccess(peer_device_id));
            }
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_multiply1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = 0;
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i%s1_n]*s2[i%s2_n];
            }
        }

        __global__ void _cuda_multiply1_add_to(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i%s1_n]*s2[i%s2_n];
            }
        }

        void multiply (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {

            DLIB_CASSERT(dest.k() == src1.k() && src1.k() == src2.k() &&
                dest.nr() == src1.nr() && src1.nr() == src2.nr() &&
                dest.nc() == src1.nc() && src1.nc() == src2.nc() ,"");
            const long MD = std::max(std::max(dest.num_samples(),src1.num_samples()),src2.num_samples());
            DLIB_CASSERT((dest.num_samples()==1 || dest.num_samples()==MD) &&
                (src1.num_samples()==1 || src1.num_samples()==MD) &&
                (src2.num_samples()==1 || src2.num_samples()==MD) ,"");

            if (dest.size() == 0)
                return;

            const size_t max_size = std::max(std::max(dest.size(),src1.size()),src2.size());
            const auto d = dest.host();
            const auto s1 = src1.host();
            const auto s2 = src2.host();
            if (dest.size() == src1.size() && src1.size() == src2.size())
            {
                if (add_to)
                    launch_kernel(_cuda_multiply1_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
                else
                    launch_kernel(_cuda_multiply1,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
            }
            else if (dest.num_samples() == 1)
            {
                if (add_to)
                    launch_kernel(_cuda_multiply2_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
                else
                    launch_kernel(_cuda_multiply2,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
            }
            else
            {
                if (add_to)
                    launch_kernel(_cuda_multiply3_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
                else
                    launch_kernel(_cuda_multiply3,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
            }
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_multiply_conv(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = s1[i]*s2[k];
            }
        }

        __global__ void _cuda_multiply_conv2(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            // zero initialize d before we begin.
            for (auto i : grid_stride_range(0, ks))
                d[i] = 0;
            __syncthreads();

            // loop over all the image planes
            for (auto i : grid_stride_range_y(0, n))
            {
                // sum all the elements in the i-th image plane
                float temp = 0;
                for (auto j : grid_stride_range(i*bs, (i+1)*bs))
                    temp += s1[j]*s2[j];
                auto k = i%ks;
                // and store the sum into d[k]
                warp_reduce_atomic_add(d[k], temp);
            }
        }

        __global__ void _cuda_multiply_conv_add_to(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] += s1[i]*s2[k];
            }
        }

        __global__ void _cuda_multiply_conv2_add_to(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            // loop over all the image planes
            for (auto i : grid_stride_range_y(0, n))
            {
                // sum all the elements in the i-th image plane
                float temp = 0;
                for (auto j : grid_stride_range(i*bs, (i+1)*bs))
                    temp += s1[j]*s2[j];
                auto k = i%ks;
                // and store the sum into d[k]
                warp_reduce_atomic_add(d[k], temp);
            }
        }


        void multiply_conv (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (have_same_dimensions(dest,src1))
            {
                DLIB_CASSERT(src2.num_samples() == 1 && src2.nr() == 1 && src2.nc() == 1 && src2.k() == src1.k(),"");
                if (dest.size() == 0)
                    return;

                if (add_to)
                    launch_kernel(_cuda_multiply_conv_add_to,max_jobs(dest.size()),
                        dest.device(), src1.device(), src1.size(), src2.device(), src1.nr()*src1.nc(), src1.k());
                else
                    launch_kernel(_cuda_multiply_conv,max_jobs(dest.size()),
                        dest.device(), src1.device(), src1.size(), src2.device(), src1.nr()*src1.nc(), src1.k());
            }
            else
            {
                DLIB_CASSERT(have_same_dimensions(src1,src2),"");
                DLIB_CASSERT(dest.num_samples() == 1 && dest.nr() == 1 && dest.nc() == 1 && dest.k() == src1.k(),"");
                if (dest.size() == 0)
                    return;

                dim3 blocks(10,1);
                dim3 threads(32,32); // x size must be 32 because we are using warp_reduce_atomic_add() in the kernel.
                if (add_to)
                    _cuda_multiply_conv2_add_to<<<blocks,threads>>>(
                        dest.device(), src1.device(), src1.num_samples()*src1.k(), src2.device(), src1.nr()*src1.nc(), src1.k());
                else
                    _cuda_multiply_conv2<<<blocks,threads>>>(
                        dest.device(), src1.device(), src1.num_samples()*src1.k(), src2.device(), src1.nr()*src1.nc(), src1.k());
            }

        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_add1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]+s2[i];
            }
        }

        __global__ void _cuda_add2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1+v2;
            }
        }

        void add (
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                launch_kernel(_cuda_add1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                // Otherwise, do the more complex version with bounds checking.
                launch_kernel(_cuda_add2,max_jobs(dest.size()),
                            dest.device(), src1.device(), src2.device(), 
                            dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                            src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                            src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                            );
            }

        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform1(float* d, const float* s, size_t n, float A, float B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s[i] + B;
            }
        }

        __global__ void _cuda_affine_transform1_0(float* d, const float* s, size_t n, float A)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s[i];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const float A,
            const float B
        )
        {
            DLIB_CASSERT(dest.size()==src.size(),"");
            if (B != 0)
                launch_kernel(_cuda_affine_transform1,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A, B);
            else
                launch_kernel(_cuda_affine_transform1_0,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A);
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const float A
        )
        {
            DLIB_CASSERT(dest.size()==src.size(),"");
            launch_kernel(_cuda_affine_transform1_0,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform4(float* d, const float* s1, const float* s2, size_t n, float A, float B, float C)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C;
            }
        }

        __global__ void _cuda_affine_transform4_0(float* d, const float* s1, const float* s2, size_t n, float A, float B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            if (C != 0)
                launch_kernel(_cuda_affine_transform4,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B, C);
            else
                launch_kernel(_cuda_affine_transform4_0,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B);
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const float A,
            const float B
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            launch_kernel(_cuda_affine_transform4_0,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_scaled(float* d, const float* s, size_t n, float scale)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += scale*s[i]; 
            }
        }

        void add_scaled(
            tensor& dest,
            const float scale,
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.size()==src.size(),"");
            launch_kernel(_cuda_add_scaled,max_jobs(dest.size()),dest.device(), src.device(), dest.size(), scale);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform5(
            float* d, const float* s1, const float* s2, const float* s3, size_t n, float A, float B, float C, float D
        )
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i] + D;
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C,
            const float D
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            DLIB_CASSERT(dest.size()==src3.size(),"");
            launch_kernel(_cuda_affine_transform5,max_jobs(dest.size()),dest.device(), src1.device(),
                src2.device(), src3.device(), dest.size(), A, B, C, D);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_range(
            float* d, const float* s1, const float* s2, const float* s3, size_t begin, size_t end, float A, float B, float C
        )
        {
            for (auto i : grid_stride_range(begin, end))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i];
            }
        }


        void affine_transform_range(
            size_t begin,
            size_t end,
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            DLIB_CASSERT(dest.size()==src3.size(),"");
            DLIB_CASSERT(begin <= end && end <= dest.size(),"");
            launch_kernel(_cuda_affine_transform_range,max_jobs(end-begin),
                dest.device(), src1.device(),
                src2.device(), src3.device(), begin, end, A, B, C);
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform2(float* d, const float* s, size_t n, const float* A, const float* B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i]*s[i] + B[i];
            }
        }
        __global__ void _cuda_affine_transform3(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i%bs]*s[i] + B[i%bs];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src),"");
            DLIB_CASSERT(
                  ((A.num_samples()==1 && B.num_samples()==1) ||
                  (A.num_samples()==src.num_samples() && B.num_samples()==src.num_samples())) &&
                  A.nr()==B.nr() && B.nr()==src.nr() &&
                  A.nc()==B.nc() && B.nc()==src.nc() &&
                  A.k() ==B.k()  && B.k()==src.k(),"");

            if (A.num_samples() == 1)
            {
                launch_kernel(_cuda_affine_transform3,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A.device(), B.device(), A.size());
            }
            else
            {
                launch_kernel(_cuda_affine_transform2,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A.device(), B.device());
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_compute_adam_update(
            size_t begin,
            size_t end,
            float* s,
            float* m,
            float* v,
            const float alpha,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const float* params,
            const float* params_grad
        )
        {
            const float eps = 1e-8;
            // The loop is equivalent to doing this:
            //   m = momentum1*m + (1-momentum1)    *   (weight_decay*params + params_grad);
            //   v = momentum2*v + (1-momentum2)*squared(weight_decay*params + params_grad);
            //   s = -alpha*m/(sqrt(v) + eps);
            for (auto i : grid_stride_range(begin, end))
            {
                float g = (weight_decay*params[i] + params_grad[i]);
                m[i] = momentum1*m[i] + (1-momentum1)*g;
                v[i] = momentum2*v[i] + (1-momentum2)*g*g;
                s[i] = -alpha*m[i]/(std::sqrt(v[i]) + eps);
            }
        }

        void compute_adam_update (
            size_t begin,
            size_t end,
            tensor& s,
            tensor& m,
            tensor& v,
            const float t,
            const float learning_rate,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const tensor& params,
            const tensor& params_grad
        )
        {
            DLIB_CASSERT(s.size() == m.size() &&
                         s.size() == v.size() &&
                         s.size() == params.size() &&
                         s.size() == params_grad.size(),"");
            DLIB_CASSERT(begin <= end && end <= params.size(),"");
            const float alpha = learning_rate*std::sqrt(1-std::pow(momentum2,t))/(1-std::pow(momentum1, t));

            launch_kernel(_cuda_compute_adam_update,max_jobs(end-begin),
                    begin, end, s.device(), m.device(), v.device(), alpha, weight_decay,
                    momentum1, momentum2, params.device(), params_grad.device());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_conv(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = A[k]*s[i] + B[k];
            }
        }

        void affine_transform_conv(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src),"");
            DLIB_CASSERT(have_same_dimensions(A, B),"");
            DLIB_CASSERT(A.num_samples() == 1 && A.nr() == 1 && A.nc() == 1 && A.k() == src.k(),"");

            launch_kernel(_cuda_affine_transform_conv,max_jobs(dest.size()),
                    dest.device(), src.device(), src.size(), A.device(), B.device(), src.nr()*src.nc(), src.k());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _add_bias_gradient(float* out, const float* in, size_t n, size_t total_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                out[i] = in[i];
                for (size_t j = i+n; j < total_n; j+=n)
                    out[i] += in[j];
            }
        }

        void assign_bias_gradient (
            tensor& grad,
            const tensor& gradient_input
        )
        {
            DLIB_CASSERT(
                  grad.num_samples() == 1 &&
                  gradient_input.k() == grad.k() &&
                  gradient_input.nr() == grad.nr() &&
                  gradient_input.nc() == grad.nc() &&
                  gradient_input.size() > 0,"");

            launch_kernel(_add_bias_gradient,max_jobs(grad.size()),grad.device(), gradient_input.device(), grad.size(), gradient_input.size());
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_threshold(float* d, size_t n, float thresh)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = d[i]>thresh ? 1:0;
            }
        }

        void threshold (
            tensor& data,
            float thresh
        )
        {
            launch_kernel(_cuda_threshold,max_jobs(data.size()),data.device(), data.size(), thresh);
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_dot(const float* a, const float* b, size_t n, float* result)
        {
            // Parallel sum everything into local temp variables.
            float temp = 0;
            for(auto i : grid_stride_range(0, n))
                temp += a[i]*b[i];

            // Then do the warp reduce add thing to merge into one output value.
            warp_reduce_atomic_add(*result, temp);
        }


        void dot (
            const tensor& a,
            const tensor& b,
            tensor& result,
            size_t idx
        )
        {
            DLIB_CASSERT(a.size() == b.size(), "");
            DLIB_CASSERT(idx < result.size(), "");

            launch_kernel(_cuda_dot, max_jobs(a.size()), a.device(), b.device(), a.size(), result.device()+idx);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_prelu(const float* s, float* d, size_t n, const float* pp)
        {
            const float p = *pp;
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    d[i] = s[i];
                else
                    d[i] = p*s[i];
            }
        }

        void prelu (
            tensor& dest,
            const tensor& src,
            const tensor& param
        )
        {
            launch_kernel(_cuda_prelu, max_jobs(dest.size()), 
                src.device(), dest.device(), src.size(), param.device());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_prelu_gradient(float* out, const float* s, const float* gi, size_t n, const float* pp, float* ppgrad)
        {
            const float p = *pp;
            float pgrad = 0;
            for(auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                {
                    out[i] += gi[i];
                }
                else
                {
                    out[i] += p*gi[i];
                    pgrad += gi[i]*s[i];
                }
            }

            // Then do the warp reduce add thing to merge into one output value.
            warp_reduce_atomic_add(*ppgrad, pgrad);
        }

        void prelu_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input,
            const tensor& param,
            tensor& params_grad 
        )
        {
            params_grad = 0;
            launch_kernel(_cuda_prelu_gradient, max_jobs(grad.size()), 
                grad.device(), src.device(), gradient_input.device(), grad.size(),
                param.device(), params_grad.device());
        }
        // ----------------------------------------------------------------------------------------

        void copy_tensor(
                tensor& dest,
                size_t dest_k_offset,
                const tensor& src,
                size_t src_k_offset,
                size_t count_k
        )
        {
            const size_t dest_sample_size = static_cast<size_t>(dest.nc() * dest.nr() * dest.k());
            const size_t src_sample_size = static_cast<size_t>(src.nc() * src.nr() * src.k());

            const size_t block_size = count_k * dest.nc() * dest.nr();

            DLIB_CASSERT(dest.num_samples() == src.num_samples() &&
                         dest.nc() == src.nc() && dest.nr() == src.nr(), "All sources should fit into dest tensor size");
            DLIB_CASSERT(dest.k() - dest_k_offset >= count_k, "Not enough space in dest tensor");
            DLIB_CASSERT(src.k() - src_k_offset >= count_k, "Not enough space in src tensor");

            float* dest_p = dest.device() + dest_k_offset * dest.nc() * dest.nr();
            const float* src_p = src.device() + src_k_offset * src.nc() * src.nr();;


            for (unsigned long i = 0; i < src.num_samples(); ++i)
            {
                CHECK_CUDA(hipMemcpy(dest_p, src_p, block_size * sizeof(float), hipMemcpyDeviceToDevice));

                dest_p += dest_sample_size;
                src_p  += src_sample_size;
            }
        }
    // ----------------------------------------------------------------------------------------

    }
}

