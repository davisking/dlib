#include "hip/hip_runtime.h"
// Copyright (C) 2015  Davis E. King (davis@dlib.net)
// License: Boost Software License   See LICENSE.txt for the full license.

#include "cuda_utils.h"
#include "cuda_dlib.h"
#include "cudnn_dlibapi.h"
#include <hip/hip_math_constants.h>


namespace dlib 
{ 
    namespace cuda 
    {

    // -----------------------------------------------------------------------------------

        void set_device (
            int dev
        )
        {
            CHECK_CUDA(hipSetDevice(dev));
        }

        int get_device (
        )
        {
            int dev = 0;
            CHECK_CUDA(hipGetDevice(&dev));
            return dev;
        }

        std::string get_device_name (
            int device
        )
        {
            hipDeviceProp_t props;
            CHECK_CUDA(hipGetDeviceProperties(&props, device));
            return props.name;
        }

        void set_current_device_blocking_sync(
        )
        {
            CHECK_CUDA(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        }

        int get_num_devices (
        )
        {
            int num_devices;
            CHECK_CUDA(hipGetDeviceCount(&num_devices));
            return num_devices;
        }

        bool can_access_peer (int device_id, int peer_device_id)
        {
            int can_access;
            CHECK_CUDA(hipDeviceCanAccessPeer(&can_access, device_id, peer_device_id));
            return can_access != 0;
        }
        bool can_access_peer (const tensor& device, const tensor& peer_device)
        {
            return can_access_peer(device.device_id(), peer_device.device_id());
        }

        void device_synchronize (int dev) 
        { 
            raii_set_device set_dev(dev);
            CHECK_CUDA(hipDeviceSynchronize());
        }
        void device_synchronize (const tensor& dev) { device_synchronize(dev.device_id()); }

        enable_peer_access::
        enable_peer_access(
            int device_id,
            int peer_device_id
        ) : call_disable(false), device_id(device_id), peer_device_id(peer_device_id)
        {
            raii_set_device set_dev(device_id);

            auto err = hipDeviceEnablePeerAccess(peer_device_id, 0);
            if (err == hipSuccess)
            {
                call_disable = true;
            }
            else if (err == hipErrorPeerAccessAlreadyEnabled)
            {
                // call hipGetLastError() to dispose of this error since we don't
                // care.
                auto err2 = hipGetLastError();
                if (err2 != hipErrorPeerAccessAlreadyEnabled)
                    CHECK_CUDA(err2);
            }
            else
            {
                CHECK_CUDA(err);
            }
        }


        enable_peer_access::
        ~enable_peer_access() noexcept(false)
        {
            if (call_disable)
            {
                raii_set_device set_dev(device_id);
                CHECK_CUDA(hipDeviceDisablePeerAccess(peer_device_id));
            }
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_inverse_norms(float* invnorms, const float* data, size_t nr, size_t nc, const float eps)
        {
            // initialize invnorms before we begin.
            for (auto i : grid_stride_range_y(0, nr))
                for (auto j : grid_stride_range(0, 1))
                    invnorms[i] = eps;
            __syncthreads();

            for (auto i : grid_stride_range_y(0, nr))
            {
                auto p = data + i*nc;
                float temp = 0;
                for (auto j : grid_stride_range(0, nc))
                    temp += p[j]*p[j];

                // and store the sum into invnorms[i]
                warp_reduce_atomic_add(invnorms[i], temp);
            }
            __syncthreads();

            for (auto i : grid_stride_range_y(0, nr))
                for (auto j : grid_stride_range(0, 1))
                    invnorms[i] = 1.0/std::sqrt(invnorms[i]);
        }

        void inverse_norms (
            resizable_tensor& invnorms,
            const tensor& data,
            const double eps
        )
        {
            invnorms.set_size(data.num_samples());
            launch_kernel(_cuda_inverse_norms, max_jobs(data.size()/data.num_samples(), data.num_samples()),
                invnorms.device(), data.device(), data.num_samples(), data.size()/data.num_samples(), eps);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_dot_prods(float* out, const float* lhs, const float* rhs, size_t nr, size_t nc)
        {
            // initialize out before we begin.
            for (auto i : grid_stride_range_y(0, nr))
                for (auto j : grid_stride_range(0, 1))
                    out[i] = 0;
            __syncthreads();

            for (auto i : grid_stride_range_y(0, nr))
            {
                auto l = lhs + i*nc;
                auto r = rhs + i*nc;
                float temp = 0;
                for (auto j : grid_stride_range(0, nc))
                    temp += l[j]*r[j];

                // and store the sum into out[i]
                warp_reduce_atomic_add(out[i], temp);
            }
        }

        __global__ void _cuda_dot_prods_add_to(float* out, const float* lhs, const float* rhs, size_t nr, size_t nc)
        {
            for (auto i : grid_stride_range_y(0, nr))
            {
                auto l = lhs + i*nc;
                auto r = rhs + i*nc;
                float temp = 0;
                for (auto j : grid_stride_range(0, nc))
                    temp += l[j]*r[j];

                // and store the sum into out[i]
                warp_reduce_atomic_add(out[i], temp);
            }
        }

        void dot_prods (
            resizable_tensor& out,
            const tensor& lhs,
            const tensor& rhs
        )
        {
            DLIB_CASSERT(have_same_dimensions(lhs,rhs));

            out.set_size(lhs.num_samples());
            if (out.size() == 0)
                return;

            const auto nr = lhs.num_samples();
            const auto nc = lhs.size()/lhs.num_samples();

            launch_kernel(_cuda_dot_prods, max_jobs(nc,nr), out.device_write_only(), lhs.device(), rhs.device(), nr, nc);
        }

        void dot_prods (
            bool add_to,
            tensor& out,
            const tensor& lhs,
            const tensor& rhs
        )
        {
            DLIB_CASSERT(have_same_dimensions(lhs,rhs));
            DLIB_CASSERT(out.k() == 1 && out.nr() == 1 && out.nc() == 1);
            DLIB_CASSERT(out.size() == lhs.num_samples());

            const auto nr = lhs.num_samples();
            const auto nc = lhs.size()/lhs.num_samples();

            if (add_to)
                launch_kernel(_cuda_dot_prods_add_to, max_jobs(nc,nr), out.device(), lhs.device(), rhs.device(), nr, nc);
            else
                launch_kernel(_cuda_dot_prods, max_jobs(nc,nr), out.device_write_only(), lhs.device(), rhs.device(), nr, nc);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_columns(float* out, const float* m, const float* v, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = m[j]*v[j%nc];
            }
        }

        void scale_columns (
            tensor& out,
            const tensor& m,
            const tensor& v
        )
        {
            launch_kernel(_cuda_scale_columns, max_jobs(m.size()), out.device(), m.device(), v.device(), m.num_samples(), m.size()/m.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_rows(float* out, const float* m, const float* v, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = m[j]*v[j/nc];
            }
        }

        void scale_rows (
            tensor& out,
            const tensor& m,
            const tensor& v
        )
        {
            launch_kernel(_cuda_scale_rows, max_jobs(m.size()), out.device(), m.device(), v.device(), m.num_samples(), m.size()/m.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_rows2(float* out, const float* m1, const float* m2, const float* v1, const float* v2, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = (m1[j] - m2[j]*v1[j/nc]) * v2[j/nc];
            }
        }

        __global__ void _cuda_scale_rows2_beta(const float beta, float* out, const float* m1, const float* m2, const float* v1, const float* v2, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = beta*out[j] + (m1[j] - m2[j]*v1[j/nc]) * v2[j/nc];
            }
        }

        void scale_rows2 (
            float beta, 
            tensor& out,
            const tensor& m1,
            const tensor& m2,
            const tensor& v1,
            const tensor& v2
        )
        {
            if (beta == 0)
            {
                launch_kernel(_cuda_scale_rows2, max_jobs(m1.size()), out.device(),
                    m1.device(), m2.device(), v1.device(), v2.device(), m1.num_samples(),
                    m1.size()/m1.num_samples());
            }
            else
            {
                launch_kernel(_cuda_scale_rows2_beta, max_jobs(m1.size()), beta,
                    out.device(), m1.device(), m2.device(), v1.device(), v2.device(),
                    m1.num_samples(), m1.size()/m1.num_samples());
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_exp(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::exp(src[i]);
        }

        void exp (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_exp, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_log(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::log(src[i]);
        }

        void log (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_log, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_log10(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::log10(src[i]);
        }

        void log10 (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_log10, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_multiply1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = 0;
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i%s1_n]*s2[i%s2_n];
            }
        }

        __global__ void _cuda_multiply1_add_to(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i%s1_n]*s2[i%s2_n];
            }
        }

        void multiply (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {

            DLIB_CASSERT(dest.k() == src1.k() && src1.k() == src2.k() &&
                dest.nr() == src1.nr() && src1.nr() == src2.nr() &&
                dest.nc() == src1.nc() && src1.nc() == src2.nc() );
            const long MD = std::max(std::max(dest.num_samples(),src1.num_samples()),src2.num_samples());
            DLIB_CASSERT((dest.num_samples()==1 || dest.num_samples()==MD) &&
                (src1.num_samples()==1 || src1.num_samples()==MD) &&
                (src2.num_samples()==1 || src2.num_samples()==MD) );

            if (dest.size() == 0)
                return;

            const size_t max_size = std::max(std::max(dest.size(),src1.size()),src2.size());
            const auto d = dest.host();
            const auto s1 = src1.host();
            const auto s2 = src2.host();
            if (dest.size() == src1.size() && src1.size() == src2.size())
            {
                if (add_to)
                    launch_kernel(_cuda_multiply1_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
                else
                    launch_kernel(_cuda_multiply1,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
            }
            else if (dest.num_samples() == 1)
            {
                if (add_to)
                    launch_kernel(_cuda_multiply2_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
                else
                    launch_kernel(_cuda_multiply2,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
            }
            else
            {
                if (add_to)
                    launch_kernel(_cuda_multiply3_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
                else
                    launch_kernel(_cuda_multiply3,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
            }
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_multiply_conv(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = s1[i]*s2[k];
            }
        }

        __global__ void _cuda_multiply_conv2(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            // zero initialize d before we begin.
            for (auto i : grid_stride_range_y(0, ks))
                for (auto j : grid_stride_range(0, 1))
                    d[i] = 0;
            __syncthreads();

            // loop over all the image planes
            for (auto i : grid_stride_range_y(0, n))
            {
                // sum all the elements in the i-th image plane
                float temp = 0;
                for (auto j : grid_stride_range(i*bs, (i+1)*bs))
                    temp += s1[j]*s2[j];
                auto k = i%ks;
                // and store the sum into d[k]
                warp_reduce_atomic_add(d[k], temp);
            }
        }

        __global__ void _cuda_multiply_conv_add_to(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] += s1[i]*s2[k];
            }
        }

        __global__ void _cuda_multiply_conv2_add_to(float* d, const float* s1, size_t n, const float* s2, size_t bs, size_t ks)
        {
            // loop over all the image planes
            for (auto i : grid_stride_range_y(0, n))
            {
                // sum all the elements in the i-th image plane
                float temp = 0;
                for (auto j : grid_stride_range(i*bs, (i+1)*bs))
                    temp += s1[j]*s2[j];
                auto k = i%ks;
                // and store the sum into d[k]
                warp_reduce_atomic_add(d[k], temp);
            }
        }


        void multiply_conv (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (have_same_dimensions(dest,src1))
            {
                DLIB_CASSERT(src2.num_samples() == 1 && src2.nr() == 1 && src2.nc() == 1 && src2.k() == src1.k());
                if (dest.size() == 0)
                    return;

                if (add_to)
                    launch_kernel(_cuda_multiply_conv_add_to,max_jobs(dest.size()),
                        dest.device(), src1.device(), src1.size(), src2.device(), src1.nr()*src1.nc(), src1.k());
                else
                    launch_kernel(_cuda_multiply_conv,max_jobs(dest.size()),
                        dest.device(), src1.device(), src1.size(), src2.device(), src1.nr()*src1.nc(), src1.k());
            }
            else
            {
                DLIB_CASSERT(have_same_dimensions(src1,src2));
                DLIB_CASSERT(dest.num_samples() == 1 && dest.nr() == 1 && dest.nc() == 1 && dest.k() == src1.k());
                if (dest.size() == 0)
                    return;


                const auto bs = src1.nr()*src1.nc();
                const auto n = src1.num_samples()*src1.k();
                if (add_to)
                    launch_kernel(_cuda_multiply_conv2_add_to, max_jobs(bs,n),
                        dest.device(), src1.device(), n, src2.device(), bs, src1.k());
                else
                    launch_kernel(_cuda_multiply_conv2, max_jobs(bs,n),
                        dest.device(), src1.device(), n, src2.device(), bs, src1.k());
            }

        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_scale_channels_add_to(float* d, const float* src, size_t n, const float* scales, size_t bs)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = i/bs;
                d[i] += src[i]*scales[k];
            }
        }

        __global__ void _cuda_scale_channels(float* d, const float* src, size_t n, const float* scales, size_t bs)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = i/bs;
                d[i] = src[i]*scales[k];
            }
        }

        void scale_channels (
            bool add_to,
            tensor& dest,
            const tensor& src,
            const tensor& scales
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest,src) && 
                         scales.num_samples() == src.num_samples() &&
                         scales.k()           == src.k() &&
                         scales.nr()          == 1 &&
                         scales.nc()          == 1 );

            if (dest.size() == 0)
                return;

            if (add_to)
                launch_kernel(_cuda_scale_channels_add_to,max_jobs(dest.size()),
                    dest.device(), src.device(), src.size(), scales.device(), src.nr()*src.nc());
            else
                launch_kernel(_cuda_scale_channels,max_jobs(dest.size()),
                    dest.device_write_only(), src.device(), src.size(), scales.device(), src.nr()*src.nc());
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_mult1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]*s2[i];
            }
        }

        __global__ void _cuda_mult1_add_to(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i]*s2[i];
            }
        }

        __global__ void _cuda_mult2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1*v2;
            }
        }

        __global__ void _cuda_mult2_add_to(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] += v1*v2;
            }
        }

        void multiply_zero_padded (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                if (add_to)
                    launch_kernel(_cuda_mult1_add_to,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
                else
                    launch_kernel(_cuda_mult1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                if (add_to)
                {
                    // Otherwise, do the more complex version with bounds checking.
                    launch_kernel(_cuda_mult2_add_to,max_jobs(dest.size()),
                                dest.device(), src1.device(), src2.device(), 
                                dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                                src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                                src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                                );
                }
                else
                {
                    // Otherwise, do the more complex version with bounds checking.
                    launch_kernel(_cuda_mult2,max_jobs(dest.size()),
                                dest.device(), src1.device(), src2.device(), 
                                dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                                src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                                src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                                );
                }
            }
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_add1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]+s2[i];
            }
        }

        __global__ void _cuda_add2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1+v2;
            }
        }

        void add (
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                launch_kernel(_cuda_add1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                // Otherwise, do the more complex version with bounds checking.
                launch_kernel(_cuda_add2,max_jobs(dest.size()),
                            dest.device(), src1.device(), src2.device(), 
                            dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                            src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                            src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                            );
            }

        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform1(float* d, const float* s, size_t n, float A, float B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s[i] + B;
            }
        }

        __global__ void _cuda_affine_transform1_0(float* d, const float* s, size_t n, float A)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s[i];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const float A,
            const float B
        )
        {
            DLIB_CASSERT(dest.size()==src.size());
            if (B != 0)
                launch_kernel(_cuda_affine_transform1,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A, B);
            else
                launch_kernel(_cuda_affine_transform1_0,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A);
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const float A
        )
        {
            DLIB_CASSERT(dest.size()==src.size());
            launch_kernel(_cuda_affine_transform1_0,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_rect(
            float* d, 
            const float* s1, 
            const float* s2, 
            const float* s3, 
            float A, 
            float B,
            float C,
            size_t start_idx,
            size_t n, 
            size_t rect_nc,
            size_t total_nc
        )
        {
            for (auto i : grid_stride_range(0, n))
            {
                size_t r = i/rect_nc;
                size_t c = i%rect_nc;
                size_t idx = r*total_nc + c + start_idx;
                d[idx] = A*s1[idx] + B*s2[idx] + C*s3[idx];
            }
        }

        void affine_transform(
            const rectangle& rect,
            tensor& dest, 
            const tensor& src1, 
            const tensor& src2, 
            const tensor& src3, 
            float A, 
            float B,
            float C
        )
        {
            DLIB_CASSERT(dest.size() == src1.size());
            DLIB_CASSERT(dest.size() == src2.size());
            DLIB_CASSERT(dest.size() == src3.size());
            DLIB_CASSERT(dest.num_samples() == src1.num_samples());
            DLIB_CASSERT(dest.num_samples() == src2.num_samples());
            DLIB_CASSERT(dest.num_samples() == src3.num_samples());
            DLIB_CASSERT(rectangle(0,0, dest.size()/dest.num_samples()-1, dest.num_samples()-1).contains(rect));
            launch_kernel(_cuda_affine_transform_rect,max_jobs(rect.area()),
                dest.device(), src1.device(), src2.device(), src3.device(), A, B, C,
                rect.left() + rect.top()*(dest.size()/dest.num_samples()),
                rect.area(),
                rect.width(),
                dest.size()/dest.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform4(float* d, const float* s1, const float* s2, size_t n, float A, float B, float C)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C;
            }
        }

        __global__ void _cuda_affine_transform4_0(float* d, const float* s1, const float* s2, size_t n, float A, float B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size());
            DLIB_CASSERT(dest.size()==src2.size());
            if (C != 0)
                launch_kernel(_cuda_affine_transform4,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B, C);
            else
                launch_kernel(_cuda_affine_transform4_0,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B);
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const float A,
            const float B
        )
        {
            DLIB_CASSERT(dest.size()==src1.size());
            DLIB_CASSERT(dest.size()==src2.size());
            launch_kernel(_cuda_affine_transform4_0,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), dest.size(), A, B);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_scaled(float* d, const float* s, size_t n, float scale)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += scale*s[i]; 
            }
        }

        void add_scaled(
            tensor& dest,
            const float scale,
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.size()==src.size());
            launch_kernel(_cuda_add_scaled,max_jobs(dest.size()),dest.device(), src.device(), dest.size(), scale);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_cv_to_all_columns(float beta, float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = beta*dest[i] + alpha*src[i/stride];
            }
        }

        __global__ void _cuda_add_cv_to_all_columns_no_beta(float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = alpha*src[i/stride];
            }
        }

        void add_cv_to_all_columns(
            float beta, 
            tensor& dest, 
            float alpha, 
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.num_samples() == src.num_samples() && src.num_samples() == src.size());
            if (beta == 0)
                launch_kernel(_cuda_add_cv_to_all_columns_no_beta, max_jobs(dest.size()), dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
            else
                launch_kernel(_cuda_add_cv_to_all_columns, max_jobs(dest.size()), beta, dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform5(
            float* d, const float* s1, const float* s2, const float* s3, size_t n, float A, float B, float C, float D
        )
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i] + D;
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C,
            const float D
        )
        {
            DLIB_CASSERT(dest.size()==src1.size());
            DLIB_CASSERT(dest.size()==src2.size());
            DLIB_CASSERT(dest.size()==src3.size());
            launch_kernel(_cuda_affine_transform5,max_jobs(dest.size()),dest.device(), src1.device(),
                src2.device(), src3.device(), dest.size(), A, B, C, D);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_range(
            float* d, const float* s1, const float* s2, const float* s3, size_t begin, size_t end, float A, float B, float C
        )
        {
            for (auto i : grid_stride_range(begin, end))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i];
            }
        }


        void affine_transform_range(
            size_t begin,
            size_t end,
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size());
            DLIB_CASSERT(dest.size()==src2.size());
            DLIB_CASSERT(dest.size()==src3.size());
            DLIB_CASSERT(begin <= end && end <= dest.size());
            launch_kernel(_cuda_affine_transform_range,max_jobs(end-begin),
                dest.device(), src1.device(),
                src2.device(), src3.device(), begin, end, A, B, C);
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform2(float* d, const float* s, size_t n, const float* A, const float* B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i]*s[i] + B[i];
            }
        }
        __global__ void _cuda_affine_transform3(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i%bs]*s[i] + B[i%bs];
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src));
            DLIB_CASSERT(
                  ((A.num_samples()==1 && B.num_samples()==1) ||
                  (A.num_samples()==src.num_samples() && B.num_samples()==src.num_samples())));
            DLIB_CASSERT(
                  A.nr()==B.nr() && B.nr()==src.nr() &&
                  A.nc()==B.nc() && B.nc()==src.nc() &&
                  A.k() ==B.k()  && B.k()==src.k(),
                  "\nA.nr(): " << A.nr() << "\nB.nr(): " << B.nr() << "\nsrc.nr(): " << src.nr()
                  <<"\nA.nc(): " << A.nc() << "\nB.nc(): " << B.nc() << "\nsrc.nc(): " << src.nc()
                  <<"\nA.k(): " << A.k() << "\nB.k(): " << B.k() << "\nsrc.k(): " << src.k()
                  );

            if (A.num_samples() == 1)
            {
                launch_kernel(_cuda_affine_transform3,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A.device(), B.device(), A.size());
            }
            else
            {
                launch_kernel(_cuda_affine_transform2,max_jobs(dest.size()),dest.device(), src.device(), src.size(), A.device(), B.device());
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_compute_adam_update(
            size_t begin,
            size_t end,
            float* s,
            float* m,
            float* v,
            const float alpha,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const float* params,
            const float* params_grad
        )
        {
            const float eps = 1e-8;
            // The loop is equivalent to doing this:
            //   m = momentum1*m + (1-momentum1)    *   (weight_decay*params + params_grad);
            //   v = momentum2*v + (1-momentum2)*squared(weight_decay*params + params_grad);
            //   s = -alpha*m/(sqrt(v) + eps);
            for (auto i : grid_stride_range(begin, end))
            {
                float g = (weight_decay*params[i] + params_grad[i]);
                m[i] = momentum1*m[i] + (1-momentum1)*g;
                v[i] = momentum2*v[i] + (1-momentum2)*g*g;
                s[i] = -alpha*m[i]/(std::sqrt(v[i]) + eps);
            }
        }

        void compute_adam_update (
            size_t begin,
            size_t end,
            tensor& s,
            tensor& m,
            tensor& v,
            const float t,
            const float learning_rate,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const tensor& params,
            const tensor& params_grad
        )
        {
            DLIB_CASSERT(s.size() == m.size() &&
                         s.size() == v.size() &&
                         s.size() == params.size() &&
                         s.size() == params_grad.size());
            DLIB_CASSERT(begin <= end && end <= params.size());
            const float alpha = learning_rate*std::sqrt(1-std::pow(momentum2,t))/(1-std::pow(momentum1, t));

            launch_kernel(_cuda_compute_adam_update,max_jobs(end-begin),
                    begin, end, s.device(), m.device(), v.device(), alpha, weight_decay,
                    momentum1, momentum2, params.device(), params_grad.device());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_conv(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = A[k]*s[i] + B[k];
            }
        }

        void affine_transform_conv(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src));
            DLIB_CASSERT(have_same_dimensions(A, B));
            DLIB_CASSERT(A.num_samples() == 1 && A.nr() == 1 && A.nc() == 1 && A.k() == src.k());

            launch_kernel(_cuda_affine_transform_conv,max_jobs(dest.size()),
                    dest.device(), src.device(), src.size(), A.device(), B.device(), src.nr()*src.nc(), src.k());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _add_bias_gradient(float* out, const float* in, size_t n, size_t total_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                out[i] = in[i];
                for (size_t j = i+n; j < total_n; j+=n)
                    out[i] += in[j];
            }
        }

        void assign_bias_gradient (
            tensor& grad,
            const tensor& gradient_input
        )
        {
            DLIB_CASSERT(
                  grad.num_samples() == 1 &&
                  gradient_input.k() == grad.k() &&
                  gradient_input.nr() == grad.nr() &&
                  gradient_input.nc() == grad.nc() &&
                  gradient_input.size() > 0);

            launch_kernel(_add_bias_gradient,max_jobs(grad.size()),grad.device(), gradient_input.device(), grad.size(), gradient_input.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _set_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] = val;
        }

        void set_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_set_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _scale_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] *= val;
        }

        void scale_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_scale_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_threshold(float* d, size_t n, float thresh)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = d[i]>thresh ? 1:0;
            }
        }

        void threshold (
            tensor& data,
            float thresh
        )
        {
            launch_kernel(_cuda_threshold,max_jobs(data.size()),data.device(), data.size(), thresh);
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_dot(const float* a, const float* b, size_t n, float* result)
        {
            // Parallel sum everything into local temp variables.
            float temp = 0;
            for(auto i : grid_stride_range(0, n))
                temp += a[i]*b[i];

            // Then do the warp reduce add thing to merge into one output value.
            warp_reduce_atomic_add(*result, temp);
        }


        void dot (
            const tensor& a,
            const tensor& b,
            tensor& result,
            size_t idx
        )
        {
            DLIB_CASSERT(a.size() == b.size());
            DLIB_CASSERT(idx < result.size());

            launch_kernel(_cuda_dot, max_jobs(a.size()), a.device(), b.device(), a.size(), result.device()+idx);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_prelu(const float* s, float* d, size_t n, const float* pp)
        {
            const float p = *pp;
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    d[i] = s[i];
                else
                    d[i] = p*s[i];
            }
        }

        void prelu (
            tensor& dest,
            const tensor& src,
            const tensor& param
        )
        {
            launch_kernel(_cuda_prelu, max_jobs(dest.size()), 
                src.device(), dest.device(), src.size(), param.device());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_prelu_gradient(float* out, const float* s, const float* gi, size_t n, const float* pp, float* ppgrad)
        {
            const float p = *pp;
            float pgrad = 0;
            for(auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                {
                    out[i] += gi[i];
                }
                else
                {
                    out[i] += p*gi[i];
                    pgrad += gi[i]*s[i];
                }
            }

            // Then do the warp reduce add thing to merge into one output value.
            warp_reduce_atomic_add(*ppgrad, pgrad);
        }

        void prelu_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input,
            const tensor& param,
            tensor& params_grad 
        )
        {
            params_grad = 0;
            launch_kernel(_cuda_prelu_gradient, max_jobs(grad.size()), 
                grad.device(), src.device(), gradient_input.device(), grad.size(),
                param.device(), params_grad.device());
        }
    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_leaky_relu(const float* s, float* d, size_t n, const float alpha)
        {
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    d[i] = s[i];
                else
                    d[i] = alpha * s[i];
            }
        }

        void leaky_relu(
            tensor& dest,
            const tensor &src,
            const float alpha
        )
        {
            launch_kernel(_cuda_leaky_relu, max_jobs(dest.size()),
                src.device(), dest.device(), src.size(), alpha);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_leaky_relu_gradient_inplace(float* out, const float* s, const float* gi, size_t n, const float alpha)
        {
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    out[i] = gi[i];
                else
                    out[i] = alpha * gi[i];
            }
        }

        __global__ void _cuda_leaky_relu_gradient(float* out, const float* s, const float* gi, size_t n, const float alpha)
        {
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    out[i] += gi[i];
                else
                    out[i] += alpha * gi[i];
            }
        }

        void leaky_relu_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input,
            const float alpha
        )
        {
            float* out = grad.device();
            const float* gi = gradient_input.device();
            if (out == gi)
            {
                launch_kernel(_cuda_leaky_relu_gradient_inplace, max_jobs(grad.size()),
                    out, src.device(), gi, grad.size(), alpha);
            }
            else
            {
                launch_kernel(_cuda_leaky_relu_gradient, max_jobs(grad.size()),
                    out, src.device(), gi, grad.size(), alpha);
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_mish(const float* s, float* d, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                const auto e = std::exp(s[i]);
                const auto delta = 2*e + e*e + 2;
                d[i] = s[i] - 2*s[i]/delta;
            }
        }

        void mish (
            tensor& dest,
            const tensor& src
        )
        {
            launch_kernel(_cuda_mish, max_jobs(dest.size()), src.device(), dest.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __device__ float mish_compute_gradient(float x)
        {
            if (x >= 8)
                return 1.f;
            if (x <= -8)
                return 0.f;

            const auto e = std::exp(x);
            const auto delta = 2*e + e*e + 2;
            const auto omega = 4*(x + 1) + 4*e*e + e*e*e + e*(4*x + 6);
            return e*omega/(delta*delta);
        }

        __global__ void _cuda_mish_gradient_inplace(float* out, const float* s, const float* gi, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] = gi[i]*mish_compute_gradient(s[i]);
        }

        __global__ void _cuda_mish_gradient(float* out, const float* s, const float* gi, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] += gi[i]*mish_compute_gradient(s[i]);
        }

        void mish_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input
        )
        {
            float* out = grad.device();
            const float* gi = gradient_input.device();
            if (out == gi)
                launch_kernel(_cuda_mish_gradient_inplace, max_jobs(grad.size()), out, src.device(), gi, grad.size());
            else
                launch_kernel(_cuda_mish_gradient, max_jobs(grad.size()), out, src.device(), gi, grad.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_gelu(const float* s, float* d, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s[i] * normcdf(s[i]);
            }
        }

        void gelu (
            tensor& dest,
            const tensor& src
        )
        {
            launch_kernel(_cuda_gelu, max_jobs(dest.size()), src.device(), dest.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __device__ float gelu_compute_gradient(float x)
        {
                const float beta = 1.0f / HIP_SQRT_2PI;
                const float cdf = normcdf(x);
                const float pdf = beta*std::exp(-0.5f*x*x);
                return cdf + x * pdf;
        }

        __global__ void _cuda_gelu_gradient_inplace(float* out, const float* s, const float* gi, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] = gi[i]*gelu_compute_gradient(s[i]);
        }

        __global__ void _cuda_gelu_gradient(float* out, const float* s, const float* gi, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] += gi[i]*gelu_compute_gradient(s[i]);
        }

        void gelu_gradient (
            tensor& grad,
            const tensor& src,
            const tensor& gradient_input
        )
        {
            float* out = grad.device();
            const float* gi = gradient_input.device();
            if (out == gi)
                launch_kernel(_cuda_gelu_gradient_inplace, max_jobs(grad.size()), out, src.device(), gi, grad.size());
            else
                launch_kernel(_cuda_gelu_gradient, max_jobs(grad.size()), out, src.device(), gi, grad.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_resize_bilinear(size_t dsize, size_t dchan_size, size_t dnc, float* d, 
                                              size_t schan_size, int snr, int snc, const float* s, 
                                              const float x_scale, const float y_scale)
        {
            for(auto i : grid_stride_range(0, dsize)) 
            {
                const int idx = i%dchan_size;
                const int channel = i/dchan_size;
                const int sidx = channel*schan_size;
                const int r = idx/dnc;
                const int c = idx%dnc;

                const float y = r*y_scale;
                const int top    = static_cast<int>(::floorf(y));
                const int bottom = ::min(top+1, snr-1);
                const float tb_frac = y - top;

                const float x = c*x_scale;
                const int left   = static_cast<int>(::floorf(x));
                const int right  = ::min(left+1, snc-1);
                const float lr_frac = x - left;

                float tl = s[sidx+top*snc+left];
                float tr = s[sidx+top*snc+right];
                float bl = s[sidx+bottom*snc+left];
                float br = s[sidx+bottom*snc+right];

                float temp = (1-tb_frac)*((1-lr_frac)*tl + lr_frac*tr) + 
                    tb_frac*((1-lr_frac)*bl + lr_frac*br);

                d[i] = temp;
            }
        }

        __global__ void _cuda_resize_bilinear_strided(size_t dsize, size_t dchan_size, size_t dnc, float* d, 
                                              size_t schan_size, int snr, int snc, const float* s, 
                                              const float x_scale, const float y_scale, 
                                              size_t dest_row_stride, size_t src_row_stride, size_t dest_chan_size_strided
                                              )
        {
            for(auto i : grid_stride_range(0, dsize)) 
            {
                const int idx = i%dchan_size;
                const int channel = i/dchan_size;
                const int sidx = channel*schan_size;
                const int r = idx/dnc;
                const int c = idx%dnc;
                const int didx = channel*dest_chan_size_strided + r*dest_row_stride+c;

                const float y = r*y_scale;
                const int top    = static_cast<int>(::floorf(y));
                const int bottom = ::min(top+1, snr-1);
                const float tb_frac = y - top;

                const float x = c*x_scale;
                const int left   = static_cast<int>(::floorf(x));
                const int right  = ::min(left+1, snc-1);
                const float lr_frac = x - left;

                float tl = s[sidx+top*src_row_stride+left];
                float tr = s[sidx+top*src_row_stride+right];
                float bl = s[sidx+bottom*src_row_stride+left];
                float br = s[sidx+bottom*src_row_stride+right];

                float temp = (1-tb_frac)*((1-lr_frac)*tl + lr_frac*tr) + 
                    tb_frac*((1-lr_frac)*bl + lr_frac*br);

                d[didx] = temp;
            }
        }

        void resize_bilinear (
            tensor& dest,
            long dest_row_stride,
            long dest_channel_stride,
            const tensor& src,
            long src_row_stride,
            long src_channel_stride
        )
        {
            DLIB_CASSERT(is_same_object(dest, src)==false);
            DLIB_CASSERT(dest.num_samples() == src.num_samples());
            DLIB_CASSERT(dest.k() == src.k());

            if (dest.size() == 0 || src.size() == 0)
                return;

            const float x_scale = (src.nc()-1)/(float)std::max<long>((dest.nc()-1),1);
            const float y_scale = (src.nr()-1)/(float)std::max<long>((dest.nr()-1),1);

            if (dest.nc() == dest_row_stride && dest.nr()*dest.nc()==dest_channel_stride &&
                src.nc()  == src_row_stride  && src.nr()*src.nc()==src_channel_stride)
            {
                launch_kernel(_cuda_resize_bilinear, 
                        dest.size(), dest.nr()*dest.nc(), dest.nc(), dest.device(),
                        src.nr()*src.nc(), src.nr(), src.nc(), src.device(),
                        x_scale, y_scale);
            }
            else
            {
                launch_kernel(_cuda_resize_bilinear_strided, 
                        dest.size(), dest.nr()*dest.nc(), dest.nc(), dest.device(),
                        src_channel_stride, src.nr(), src.nc(), src.device(),
                        x_scale, y_scale, dest_row_stride, src_row_stride, dest_channel_stride);
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_resize_bilinear_gradient(size_t dsize, size_t dchan_size, size_t dnc, const float* d, 
                                              size_t schan_size, int snr, int snc, float* s, 
                                              const float x_scale, const float y_scale)
        {
            for(auto i : grid_stride_range(0, dsize)) 
            {
                const float tmp = d[i];

                const int idx = i%dchan_size;
                const int channel = i/dchan_size;
                const int sidx = channel*schan_size;
                const int r = idx/dnc;
                const int c = idx%dnc;

                const float y = r*y_scale;
                const int top    = static_cast<int>(::floorf(y));
                const int bottom = ::min(top+1, snr-1);
                const float tb_frac = y - top;

                const float x = c*x_scale;
                const int left   = static_cast<int>(::floorf(x));
                const int right  = ::min(left+1, snc-1);
                const float lr_frac = x - left;


                atomicAdd(s+sidx+top*snc+left,     tmp*(1-tb_frac)*(1-lr_frac));
                atomicAdd(s+sidx+top*snc+right,    tmp*(1-tb_frac)*(lr_frac));
                atomicAdd(s+sidx+bottom*snc+left,  tmp*(tb_frac)*(1-lr_frac));
                atomicAdd(s+sidx+bottom*snc+right, tmp*(tb_frac)*(lr_frac));
            }
        }

        __global__ void _cuda_resize_bilinear_gradient_strided(size_t dsize, size_t dchan_size, size_t dnc, const float* d, 
                                              size_t schan_size, int snr, int snc, float* s, 
                                              const float x_scale, const float y_scale,
                                              size_t dest_row_stride, size_t src_row_stride, size_t dest_chan_size_strided
                                              )
        {
            for(auto i : grid_stride_range(0, dsize)) 
            {

                const int idx = i%dchan_size;
                const int channel = i/dchan_size;
                const int didx = channel*dest_chan_size_strided;
                const int sidx = channel*schan_size;
                const int r = idx/dnc;
                const int c = idx%dnc;

                const float tmp = d[didx + r*dest_row_stride+c];

                const float y = r*y_scale;
                const int top    = static_cast<int>(::floorf(y));
                const int bottom = ::min(top+1, snr-1);
                const float tb_frac = y - top;

                const float x = c*x_scale;
                const int left   = static_cast<int>(::floorf(x));
                const int right  = ::min(left+1, snc-1);
                const float lr_frac = x - left;


                atomicAdd(s+sidx+top*src_row_stride+left,     tmp*(1-tb_frac)*(1-lr_frac));
                atomicAdd(s+sidx+top*src_row_stride+right,    tmp*(1-tb_frac)*(lr_frac));
                atomicAdd(s+sidx+bottom*src_row_stride+left,  tmp*(tb_frac)*(1-lr_frac));
                atomicAdd(s+sidx+bottom*src_row_stride+right, tmp*(tb_frac)*(lr_frac));
            }
        }

        void resize_bilinear_gradient (
            tensor& grad,
            long grad_row_stride,
            long grad_channel_stride,
            const tensor& gradient_input,
            long gradient_input_row_stride,
            long gradient_input_channel_stride
        )
        {
            DLIB_CASSERT(is_same_object(grad, gradient_input)==false);
            DLIB_CASSERT(gradient_input.num_samples() == grad.num_samples());
            DLIB_CASSERT(gradient_input.k() == grad.k());

            if (grad.size() == 0 || gradient_input.size() == 0)
                return;

            const float x_scale = (grad.nc()-1)/(float)std::max<long>((gradient_input.nc()-1),1);
            const float y_scale = (grad.nr()-1)/(float)std::max<long>((gradient_input.nr()-1),1);

            if (grad.nc() == grad_row_stride && grad.nr()*grad.nc()==grad_channel_stride &&
                gradient_input.nc() == gradient_input_row_stride && gradient_input.nr()*gradient_input.nc()==gradient_input_channel_stride)
            {
                launch_kernel(_cuda_resize_bilinear_gradient, 
                        gradient_input.size(), gradient_input.nr()*gradient_input.nc(), gradient_input.nc(), gradient_input.device(),
                        grad.nr()*grad.nc(), grad.nr(), grad.nc(), grad.device(),
                        x_scale, y_scale);
            }
            else
            {
                launch_kernel(_cuda_resize_bilinear_gradient_strided, 
                        gradient_input.size(), gradient_input.nr()*gradient_input.nc(), gradient_input.nc(), gradient_input.device(),
                        grad_channel_stride, grad.nr(), grad.nc(), grad.device(),
                        x_scale, y_scale, gradient_input_row_stride, grad_row_stride, gradient_input_channel_stride);
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_layer_normalize(float* out, const float* s, float* m, float* v, const float* g, const float* b, float eps, size_t ns, size_t num)
        {
           // compute means and sum of squares
            for (auto n : grid_stride_range_y(0, ns))
            {
                auto p = s + n * num;
                float means = 0;
                float invstds = 0;
                for (auto i : grid_stride_range(0, num))
                {
                    means += p[i];
                    invstds += p[i] * p[i];
                }
                warp_reduce_atomic_add(m[n], means/num);
                warp_reduce_atomic_add(v[n], invstds/num);
            }
            __syncthreads();

            // compute variances
            for (auto n : grid_stride_range_y(0, ns))
            {
                for (auto i : grid_stride_range(0, 1))
                {
                    auto var = v[n] - m[n] * m[n];
                    v[n] = 1.0f / std::sqrt(var + eps);
                }
            }
            __syncthreads();

            for (auto n : grid_stride_range_y(0, ns))
            {
                for (auto i : grid_stride_range(0, num))
                {
                    const float val = (s[n*num+i]-m[n])*v[n];
                    out[n*num+i] = val*g[n]+b[n];
                }
            }
        }

        __global__ void _cuda_layer_normalize_gradient(float* out, float* gg, float* bg, const float* s, const float* gi, const float* m, const float* v, const float* g, float* dm, float* dv, float eps, size_t ns, size_t num)
        {
            for (auto n : grid_stride_range_y(0, ns))
            {
                float temp_bg = 0;
                float temp_gg = 0;
                float temp_dv = 0;
                for (auto i : grid_stride_range(0, num))
                {
                    auto idx = n*num+i;
                    const float x_hat = (s[idx] - m[n])*v[n];
                    temp_bg += gi[idx];
                    temp_gg += gi[idx]*x_hat;

                    const float dx = gi[idx] * g[n];
                    temp_dv += dx*(s[idx] - m[n])*-0.5*v[n]*v[n]*v[n];
                }
                warp_reduce_atomic_add(bg[n], temp_bg);
                warp_reduce_atomic_add(gg[n], temp_gg);
                warp_reduce_atomic_add(dv[n], temp_dv);
            }
            __syncthreads();

            for (auto n : grid_stride_range_y(0, ns))
            {
                float temp_dm = 0;
                for (auto i : grid_stride_range(0, num))
                {
                    auto idx = n*num+i;
                    const float dx = gi[idx]*g[n];
                    temp_dm += dx*-v[n] + dv[n] * -2*(s[idx] - m[n])/num;
                }
                warp_reduce_atomic_add(dm[n], temp_dm);
            }
            __syncthreads();

            for (auto n : grid_stride_range_y(0, ns))
            {
                for (auto i : grid_stride_range(0, num))
                {
                    auto idx = n*num+i;
                    const float dx = gi[idx]*g[n];
                    out[idx] += dx*v[n] + dv[n] * 2*(s[idx] - m[n])/num + dm[n]/num;
                }
            }
        }

        void layer_normalize (
            const double eps,
            resizable_tensor& dest,
            resizable_tensor& means,
            resizable_tensor& invstds,
            const tensor& src,
            const tensor& gamma,
            const tensor& beta
        )
        {
            const long num = src.k() * src.nr() * src.nc();
            DLIB_CASSERT(
                have_same_dimensions(gamma, beta) &&
                src.num_samples() == gamma.size() &&
                src.num_samples() == beta.size() &&
                eps > 0,
                "\ngamma.k():  " << gamma.k() <<
                "\ngamma.nr(): " << gamma.nr() <<
                "\ngamma.nc(): " << gamma.nc() <<
                "\nbeta.k():   " << beta.k() <<
                "\nbeta.nr():  " << beta.nr() <<
                "\nbeta.nc():  " << beta.nc() <<
                "\nsrc.k():   " << src.k() <<
                "\nsrc.nr():  " << src.nr() <<
                "\nsrc.nc():  " << src.nc() <<
                "\neps:  " << eps
            );

            dest.copy_size(src);
            means.set_size(src.num_samples());
            invstds.set_size(src.num_samples());
            means = 0;
            invstds = 0;
            launch_kernel(_cuda_layer_normalize, max_jobs(num, src.num_samples()), dest.device(), src.device(),
                          means.device(), invstds.device(), gamma.device(), beta.device(), eps, src.num_samples(), num);
        }

        void layer_normalize_gradient (
            const double eps,
            const tensor& gradient_input,
            const tensor& means,
            const tensor& invstds,
            const tensor& src,
            const tensor& gamma,
            tensor& src_grad,
            tensor& gamma_grad,
            tensor& beta_grad
        )
        {
            const long num = src.k() * src.nr() * src.nc();
            DLIB_CASSERT(src.num_samples() == means.size());
            DLIB_CASSERT(src.num_samples() == invstds.size());
            DLIB_CASSERT(src.num_samples() == gamma.size());
            DLIB_CASSERT(src.num_samples() == gamma_grad.size());
            DLIB_CASSERT(src.num_samples() == beta_grad.size());
            DLIB_CASSERT(have_same_dimensions(gradient_input, src));
            DLIB_CASSERT(have_same_dimensions(gradient_input, src_grad));
            DLIB_CASSERT(eps > 0);

            beta_grad = 0;
            gamma_grad = 0;
            resizable_tensor dvars, dmeans;
            dvars.copy_size(invstds);
            dmeans.copy_size(means);
            dvars = 0;
            dmeans = 0;
            launch_kernel(_cuda_layer_normalize_gradient, max_jobs(num, src.num_samples()),
                          src_grad.device(), gamma_grad.device(), beta_grad.device(), src.device(),
                          gradient_input.device(), means.device(), invstds.device(), gamma.device(),
                          dmeans.device(), dvars.device(), eps, src.num_samples(), num);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_copy_tensor_add_to (float* dest, size_t size,  const float* src,  size_t dest_stride, size_t src_stride, size_t block_size)
        {
            for(auto i : grid_stride_range(0, size)) 
            {
                size_t blk = i/block_size;
                size_t j = i%block_size;
                dest[blk*dest_stride + j] += src[blk*src_stride + j];
            }
        }

        __global__ void _cuda_copy_tensor (float* dest, size_t size,  const float* src,  size_t dest_stride, size_t src_stride, size_t block_size)
        {
            for(auto i : grid_stride_range(0, size)) 
            {
                size_t blk = i/block_size;
                size_t j = i%block_size;
                dest[blk*dest_stride + j] = src[blk*src_stride + j];
            }
        }

        void copy_tensor(
            bool add_to,
            tensor& dest,
            size_t dest_k_offset,
            const tensor& src,
            size_t src_k_offset,
            size_t count_k
        )
        {
            const size_t dest_sample_size = static_cast<size_t>(dest.nc() * dest.nr() * dest.k());
            const size_t src_sample_size = static_cast<size_t>(src.nc() * src.nr() * src.k());

            const size_t block_size = count_k * dest.nc() * dest.nr();

            DLIB_CASSERT(dest.num_samples() == src.num_samples() &&
                         dest.nc() == src.nc() && dest.nr() == src.nr(), "All sources should fit into dest tensor size");
            DLIB_CASSERT(dest.k() - dest_k_offset >= count_k, "Not enough space in dest tensor");
            DLIB_CASSERT(src.k() - src_k_offset >= count_k, "Not enough space in src tensor");

            float* dest_p = dest.device() + dest_k_offset * dest.nc() * dest.nr();
            const float* src_p = src.device() + src_k_offset * src.nc() * src.nr();;

            if (add_to)
            {
                launch_kernel(_cuda_copy_tensor_add_to, max_jobs(dest.size()), 
                              dest_p, block_size*dest.num_samples(),
                              src_p, dest_sample_size, src_sample_size, block_size);
            }
            else
            {
                launch_kernel(_cuda_copy_tensor, max_jobs(dest.size()), 
                              dest_p, block_size*dest.num_samples(),
                              src_p, dest_sample_size, src_sample_size, block_size);
            }
        }

    // ----------------------------------------------------------------------------------------

        __device__ float cuda_log1pexp(float x)
        {
            if (x <= -18)
                return std::exp(x);
            else if (-18 < x && x <= 9)
                return std::log1pf(std::exp(x));
            else if (9 < x && x <= 16)
                return x + expf(-x);
            else
                return x;
        }

        __global__ void _cuda_compute_loss_binary_log_per_pixel(float* loss_out, float* g, const float* truth, const float* out_data, size_t n, const float scale)
        {
            float loss = 0;
            for(auto i : grid_stride_range(0, n))
            {
                const float y = truth[i];

                if (y > 0.f)
                {
                    const float temp = cuda_log1pexp(-out_data[i]);
                    loss += y*temp;
                    g[i] = y*scale*(g[i]-1);
                }
                else if (y < 0.f)
                {
                    const float temp = -(-out_data[i]-cuda_log1pexp(-out_data[i]));
                    loss += -y*temp;
                    g[i] = -y*scale*g[i];
                }
                else
                {
                    g[i] = 0.f;
                }
            }

            warp_reduce_atomic_add(*loss_out, loss);
        }

    // ----------------------------------------------------------------------------------------

        __device__ float cuda_safe_log(float x, float epsilon = 1e-10)
        {
            // Prevent trying to calculate the logarithm of a very small number (let alone zero)
            if (x >= epsilon)
                return ::log(x);
            else
                return ::log(epsilon);
        }

        __global__ void _cuda_compute_loss_multiclass_log_per_pixel(float* loss_out, float* g, const uint16_t* truth, size_t n, size_t plane_size, size_t sample_size, size_t nk, uint16_t label_to_ignore, const float scale)
        {
            float loss = 0;
            for(auto i : grid_stride_range(0, n))
            {
                const size_t k = (i/plane_size)%nk;
                const size_t idx = (i%plane_size) + plane_size*(i/sample_size);

                const size_t y = truth[idx];

                if (k == y)
                {
                    loss -= cuda_safe_log(g[i]);
                    g[i] = scale*(g[i] - 1);
                }
                else if (y == label_to_ignore)
                {
                    g[i] = 0.f;
                }
                else
                {
                    g[i] = scale*g[i];
                }
            }

            warp_reduce_atomic_add(*loss_out, loss);
        }

        __global__ void _cuda_compute_loss_multiclass_log_per_pixel_weighted(float* loss_out, float* g, const uint16_t* truth, size_t n, size_t plane_size, size_t sample_size, size_t nk, const float* weights, const float scale)
        {
            float loss = 0;
            for(auto i : grid_stride_range(0, n))
            {
                const size_t k = (i/plane_size)%nk;
                const size_t idx = (i%plane_size) + plane_size*(i/sample_size);

                const size_t y = truth[idx];
                const float weight = weights[idx];

                if (k == y)
                {
                    loss -= weight*cuda_safe_log(g[i]);
                    g[i] = weight*scale*(g[i] - 1);
                }
                else
                {
                    g[i] = weight*scale*g[i];
                }
            }

            warp_reduce_atomic_add(*loss_out, loss);
        }
    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_compute_loss_mean_squared_per_channel_and_pixel(float* loss_out, float* g, const float* truth, const float* out_data, size_t n, const float scale)
        {
            float loss = 0;
            for (auto i : grid_stride_range(0, n))
            {
                const float y = truth[i];
                const float temp = y - out_data[i];
                loss += temp * temp;
                g[i] = -temp * scale;
            }
            warp_reduce_atomic_add(*loss_out, loss);
        }

    // ----------------------------------------------------------------------------------------

        void compute_loss_binary_log_per_pixel::
        do_work(
            cuda_data_ptr<float> loss_work_buffer,
            cuda_data_ptr<const float> truth_buffer,
            const tensor& subnetwork_output,
            tensor& gradient,
            double& loss
        )
        {
            CHECK_CUDA(hipMemset(loss_work_buffer, 0, sizeof(float)));
            sigmoid(gradient, subnetwork_output);

            // The loss we output is the average loss over the mini-batch, and also over each element of the matrix output.
            const double scale = 1.0 / (subnetwork_output.num_samples() * subnetwork_output.nr() * subnetwork_output.nc());

            launch_kernel(_cuda_compute_loss_binary_log_per_pixel, max_jobs(gradient.size()),
                loss_work_buffer.data(), gradient.device(), truth_buffer.data(), subnetwork_output.device(), gradient.size(), scale);

            float floss;
            dlib::cuda::memcpy(&floss, loss_work_buffer);
            loss = scale*floss;
        }

        void compute_loss_multiclass_log_per_pixel::
        do_work(
            cuda_data_ptr<float> loss_work_buffer,
            cuda_data_ptr<const uint16_t> truth_buffer,
            const tensor& subnetwork_output,
            tensor& gradient,
            double& loss
        )
        {
            CHECK_CUDA(hipMemset(loss_work_buffer, 0, sizeof(float)));
            softmax(gradient, subnetwork_output);
            static const uint16_t label_to_ignore = std::numeric_limits<uint16_t>::max();

            // The loss we output is the average loss over the mini-batch, and also over each element of the matrix output.
            const double scale = 1.0 / (subnetwork_output.num_samples() * subnetwork_output.nr() * subnetwork_output.nc());

            launch_kernel(_cuda_compute_loss_multiclass_log_per_pixel, max_jobs(gradient.size()),
                loss_work_buffer.data(), gradient.device(), truth_buffer.data(), gradient.size(), gradient.nr()*gradient.nc(), gradient.nr()*gradient.nc()*gradient.k(), gradient.k(), label_to_ignore, scale);

            float floss;
            dlib::cuda::memcpy(&floss, loss_work_buffer);
            loss = scale*floss;
        }

        void compute_loss_multiclass_log_per_pixel_weighted::
        do_work(
            cuda_data_ptr<float> loss_work_buffer,
            cuda_data_ptr<const uint16_t> truth_buffer,
            cuda_data_ptr<const float> weights_buffer,
            const tensor& subnetwork_output,
            tensor& gradient,
            double& loss
        )
        {
            CHECK_CUDA(hipMemset(loss_work_buffer, 0, sizeof(float)));
            softmax(gradient, subnetwork_output);

            // The loss we output is the average loss over the mini-batch, and also over each element of the matrix output.
            const double scale = 1.0 / (subnetwork_output.num_samples() * subnetwork_output.nr() * subnetwork_output.nc());

            launch_kernel(_cuda_compute_loss_multiclass_log_per_pixel_weighted, max_jobs(gradient.size()),
                loss_work_buffer.data(), gradient.device(), truth_buffer.data(), gradient.size(), gradient.nr()*gradient.nc(), gradient.nr()*gradient.nc()*gradient.k(), gradient.k(), weights_buffer.data(), scale);

            float floss;
            dlib::cuda::memcpy(&floss, loss_work_buffer);
            loss = scale*floss;
        }

        void compute_loss_mean_squared_per_channel_and_pixel::
        do_work(
            cuda_data_ptr<float> loss_work_buffer,
            cuda_data_ptr<const float> truth_buffer,
            const tensor& subnetwork_output,
            tensor& gradient,
            double& loss
        )
        {
            CHECK_CUDA(hipMemset(loss_work_buffer, 0, sizeof(float)));

            // The loss we output is the average loss over the mini-batch, and also over each element of the matrix output.
            const double scale = 1.0 / (subnetwork_output.num_samples() * subnetwork_output.k() * subnetwork_output.nr() * subnetwork_output.nc());

            launch_kernel(_cuda_compute_loss_mean_squared_per_channel_and_pixel , max_jobs(gradient.size()),
                loss_work_buffer.data(), gradient.device(), truth_buffer.data(), subnetwork_output.device(), gradient.size(), scale);

            float floss;
            dlib::cuda::memcpy(&floss, loss_work_buffer);
            loss = scale*floss;
        }

    // ----------------------------------------------------------------------------------------

    }
}

